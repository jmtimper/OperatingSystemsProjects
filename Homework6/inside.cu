// Elapsed Real Time for input-c4.txt:
// Elapsed Real Time for input-c5.txt:

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Simple struct for representing a circle.
typedef struct {
  int x, y;
  int r;
} Circle;

// List of all circles.
Circle *cList;

// Number of circles on our list.
int cCount = 0;

// Function run by each thread.  You're going to need to pass
// in some more parameters.
__global__ void countCircles( int n, Circle * list, int *out ) {

  // Determine a unique index for this worker.
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  // counter
  int countTemp = 0;
  
  // Make sure I actually have something to work on.
  if ( i < n ) {
    // This thread has a circle to work on.
    Circle *c = list + i;
    for ( int x = c->x - c->r; x <= c->x + c->r; x++ ){
      for ( int y = c->y - c->r; y <= c->y + c->r; y++ ){
        int dx = x - c->x;
        int dy = y - c->y;
        if ( dx * dx + dy * dy <= c->r * c->r ) {
          bool counted = false;
          for ( int k = 0; !counted && k < i; k++ ){
            Circle *u = list + k;
            int dx = x - u->x;
            int dy = y - u->y;
            if (  dx * dx + dy * dy <= u->r * u->r  )
              counted = true;
          }
          if ( !counted )
            countTemp++;
        }
      }
    }
    out[i] = countTemp;
  }
}

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message ); 
  exit( 1 );
}

// Read the list of circles.
void readCircles() {
  // Use a resizable array, increasing capacity as we read more values.
  int capacity = 10;
  cList = (Circle *) malloc( capacity * sizeof( Circle ) );

  // Read circle definitions until we can't read any more.
  Circle c;
  while ( scanf( "%d%d%d", &c.x, &c.y, &c.r ) == 3 ) {
    // Grow the array if needed.
    if ( cCount >= capacity ) {
      capacity *= 2;
      cList = (Circle *) realloc( cList, capacity * sizeof( Circle ) );
    }
    
    // Add the circle we just read to the lsit.
    cList[ cCount ] = c;
    cCount++;
  }
}

int main( ) {
  readCircles();

  // Block and grid dimensions.
  int threadsPerBlock = 250;
  // Round up to get the number of blocks we need.
  int blocksPerGrid = ( cCount + threadsPerBlock - 1 ) / threadsPerBlock;

  //copy over clist to GPU
  Circle *list = NULL;
  hipMalloc( (void **)&list, cCount * sizeof(Circle));
  hipMemcpy(list, cList, cCount * sizeof(Circle), hipMemcpyHostToDevice);
  //int array for gpu output
  int *gpuOut = NULL;
  hipMalloc( (void **)&gpuOut, cCount * sizeof(int));

  // Run our kernel on these block/grid dimensions.  You'll need to
  // pass in some more parameters.
  countCircles<<<blocksPerGrid, threadsPerBlock>>>( cCount, list , gpuOut);
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // int array of output from gpu
  int result[cCount];
  hipMemcpy(result, gpuOut, cCount * sizeof(int), hipMemcpyDeviceToHost);
  
  // Add up the total and report it.
  int total = 0;
  for(int i = 0; i < cCount; i++)
    total += result[i];
  printf( "Total: %d\n", total );

  // Free resources and reset the device.
  free( cList );
  hipFree( list );
  hipFree( gpuOut );
  hipDeviceReset();
}
